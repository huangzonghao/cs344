#include "hip/hip_runtime.h"
#include "utils.h"
struct ExtremeValues{
        float max;
        float min;
};

__global__ 
void findExtremeValues( const float* const d_data, 
                        const size_t numCols,
                       int numRows,
                        ExtremeValues * d_extreme){

//        extern __shared__ s_extreme[];
        // no need to initialize since we are not accumulate with it
        __shared__  ExtremeValues s_extreme;

        int startIdx = threadIdx.x * numCols;
        int endIdx  = startIdx + numCols; // exclusive
        ExtremeValues out;
        out.max = d_data[startIdx];
        out.min = out.max;
        float temp; // with this variable we just need to read the global memory once for each iteration instead of twice
        for ( int i = startIdx + 1; i < endIdx; ++i){
                temp = d_data[i];
                if (temp < out.min)
                       out.min = temp;
               if (temp > out.max)
                      out.max = temp; 
        }
        //s_extreme[threadIdx.x].max = out.max;
        //s_extreme[threadIdx.x].min = out.min;

        //atomicMin((float*)&(s_extreme.min), out.min);
        //atomicMax(&(s_extreme.max), out.max);
    // since both the atomicMin and atomicMax supports only the int, we should find someway else to solve the problem
       __syncthreads();
    if (threadIdx.x ==0){
        s_extreme.max = out.max;
        s_extreme.min = out.min;
    }
    __syncthreads();
    for (int i = 1; i < numRows; ++i){
        __syncthreads();
        if ( threadIdx.x == i){
            if (out.min < s_extreme.min)
                s_extreme.min = out.min;
            if (out.max > s_extreme.max)
                s_extreme.max = out.max;
        }
    }
        __syncthreads();

        if (threadIdx.x == 1){
                d_extreme->max = s_extreme.max;
                d_extreme->min = s_extreme.min;
        }

}

// take each row as a thread
__global__
void generateHisto(const float* d_lum, 
                   const int lumMin, 
                   const float lumRange, 
                   const int numBins, 
                   const int numRows,
                   const int numCols,
                   int *d_histo){

        // needed to be initialized, and the size of this shared memory is numRows * numBins
        extern int __shared__ s_histo[];
        for (int i = threadIdx.x; i < numRows * numBins; i += numRows)
        {s_histo[i] = 0;
         d_histo[i] = 0;
        }
        // note above is the general way to initialize shared or global memory

        int startIdx = threadIdx.x * numCols + 1;
        int endIdx  = startIdx + numCols; // exclusive
        int binStartIdx = threadIdx.x * numBins; // inclusive

        int binIdx = 0;
    
        for (int i = startIdx; i < endIdx; ++i){
                binIdx = binStartIdx + (d_lum[i] - lumMin) / lumRange * numBins;
                ++s_histo[binIdx];
        } 

        __syncthreads();

        // now starting merge the local bins
        // we have numRows threads and numBins Bins to deal with, that's numRows/numBins per Bin. but for simplicity, we use 2 threads per bin here
        if ( numRows >= 2 * numBins){ // in this case we have enough threads to run all the threads in parallel, and we use 2 threads for each bin
                // application of 2 threads reduce
                       if ( threadIdx.x < 2 * numBins){
                               binIdx = threadIdx.x /2;  // so frist two threads for the first bin, then the following two threads for the second bin, etc.
                               startIdx = ( binIdx + (threadIdx.x % 2) * numRows/2) * numBins;
                               endIdx =  startIdx + ((threadIdx.x + 1) % 2) * ( numRows/2 ) * numBins + (threadIdx.x % 2) * (numRows) * numBins;
                               int out = 0;
                               for ( int i = startIdx; i < endIdx; i += numBins)
                                      out += s_histo[i];
                               __syncthreads();

                               atomicAdd( &d_histo[binIdx], out);
                       } 
        }

        // because for this case we don't have enough threads to reduce all the bins in parallel, then it makes no sense for some threads to do it in parallel then waiting for the solo threads. so we just merge the following two cases together
        //else if (numRows >= numBins){ 

        //}
        else { // now less threads than bins
                float out = 0;
                for (int iBins = threadIdx.x; iBins < numBins; iBins += numRows){
                        for (int i = 0; i < numRows; ++i){
                                out += s_histo[iBins + i * numBins];
                        }
                        // one thread for one bin, no need to use atomicAdd
                        d_histo[iBins] = out;
                }

        }
}

// frist we just assume that the numBins is a number of two to some power

__global__
void bellochScan(const int * d_histo,
                 const int numBins,
                 unsigned int * d_cdf){
        // first we need to copy the data to the shared memory
        extern int __shared__ s_cdf[];
        // numBins/2 threads will be enough
        for ( size_t i = threadIdx.x; i < numBins; i += numBins/2){
                s_cdf[i] = d_histo[i];
        }

                __syncthreads(); // make sure the copying is finished
        // now starting the scan
        size_t startIdx = 0; // this is the starting index of the first thread which will be in used in this round
        size_t myIdx = 0;  // the index of the left element of the pair
        size_t parterIdx = 0; // the index of the right element of the pair
        size_t step_size = 0;
        for (step_size = 1; step_size < numBins; step_size <<= 1){
                __syncthreads(); // make sure all the threads have done the first round then move on to the next

                startIdx = step_size - 1;
                myIdx = startIdx + threadIdx.x * 2 * step_size; // which means the smaller the threadIdx.x is, the more often the thread will be used, and thread 0 will be at full run
                if ( myIdx < numBins - 1){  // when myIdx locates at the last element, we don't count that one
                        parterIdx = myIdx + step_size;
                        if (parterIdx >= numBins)
                                parterIdx = numBins - 1;
                        s_cdf[parterIdx] = s_cdf[parterIdx] + s_cdf[myIdx];
                }
        }
        // the up-sweep finishes here, now the following is down-sweep
        if ( threadIdx.x == 0)
                s_cdf[numBins - 1] = 0;
        __syncthreads();

        int temp;
        // the code is roughly the same as the above one
        for (step_size >>= 1; step_size >= 1; step_size >>= 1){
                __syncthreads();

                startIdx = step_size - 1;
                myIdx = startIdx + threadIdx.x * 2 * step_size;

                 if ( myIdx < numBins - 1){  // when myIdx locates at the last element, we don't count that one
                        parterIdx = myIdx + step_size;
                        if (parterIdx >= numBins)
                                parterIdx = numBins - 1;
                        temp  = s_cdf[parterIdx];
                        s_cdf[parterIdx] = temp + s_cdf[myIdx];
                        s_cdf[myIdx] = temp;
                }
       } 
        
        __syncthreads();

        // now copy back the result to the global memory
        for ( size_t i = threadIdx.x; i < numBins; i += numBins/2)
                d_cdf[i] = s_cdf[i];

}
void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

        //1) caculating the min and max
        ExtremeValues *d_extreme;
        checkCudaErrors(hipMalloc(&d_extreme, sizeof(ExtremeValues)));
        // here a row is a block
        findExtremeValues<<<numRows, numCols>>>( d_logLuminance, numCols, numRows, d_extreme);
        min_logLum = d_extreme->min;
        max_logLum = d_extreme->max;
        
        //2) finding the range
        float range = max_logLum - min_logLum;

        //3) generating the histogram
        // allocating global memory for the histogram
        int * d_histo;
        checkCudaErrors(hipMalloc(&d_histo, sizeof(int) * numBins));

        generateHisto<<< 1, numRows, numRows * numBins * sizeof(int) >>>( d_logLuminance, min_logLum, range, numBins, numRows, numCols, d_histo); 

        //4) the exclusive scan for cdf
        // we try to implement the Blelloch Scan here
        // numBins/2 threads are used, and a shared memory of numBins in size are allocated
        
        bellochScan<<< 1, numBins/2, numBins * sizeof(int)>>>(d_histo, numBins, d_cdf); 

}